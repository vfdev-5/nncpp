#include "hip/hip_runtime.h"
// STD
#include <cassert>
#include <iostream>

// CUDA
#include <hip/hip_cooperative_groups.h>


// NNCpp
#include "tensor.hpp"
#include "activations.hpp"
#include "cuda_tensor_wrapper.cuh"
#include "cuda_utils.cuh"


namespace cg = cooperative_groups;


namespace nncpp
{


__global__ void kerner_relu(CUDATensorWrapper input, CUDATensorWrapper output)
{    
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < input.numel())
    {
        if (input.at(i) < 0.0f)
        {            
            output.at(i) = 0.0f;
        }
        else if (input.const_data() != output.data())
        { // copy data if output is not input            
            output.at(i) = input.at(i);
        }
        
    }
}


__global__ void kerner_relu_backward(CUDATensorWrapper grad, CUDATensorWrapper input, CUDATensorWrapper output)
{    
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < input.numel())
    {
        float value = grad.at(i);
        if (input.at(i) < 0.0f)
        {            
            value = 0.0f;
        }
        output.at(i) = value;
    }
}


__global__ void kerner_sigmoid(CUDATensorWrapper input, CUDATensorWrapper output)
{    
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < input.numel())
    {
        float exp_i = expf(input.at(i));     
        output.at(i) = exp_i / (1.0f + exp_i);        
    }
}


__global__ void kerner_sigmoid_backward(CUDATensorWrapper grad, CUDATensorWrapper input, CUDATensorWrapper output)
{    
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < input.numel())
    {        
        float value = input.at(i);
        output.at(i) = -1.0f * expf(value) * expm1f(value) * grad.at(i);
    }
}


__global__ void kerner_softmax(CUDATensorWrapper input, size_t dim, CUDATensorWrapper output, CUDATensorWrapper buffer)
{   
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();

    // softmax(input, dim) = exp(input - max(input)) / sum(exp(input - max(input)), dim)
    // a) compute max(input)    
    _kernel_reduce_op(input, buffer, op_max, _atomicMax);

    cg::sync(cta); 
    // b) compute sum(exp(input - max(input)), dim)
    float max_input = buffer.at(0);
    // _kernel_reduce_op_on_dim(input, dim, )

    // c) compute exp(input - max(input)) / sum(exp(input - max(input)), dim)

}


void _elementwise_activation_inplace(Tensor & input, void kernel_func(CUDATensorWrapper, CUDATensorWrapper))
{
    assert(input.device == Device::CUDA);    
    int grid_size = setup_grid_size(input.numel(), BLOCK_SIZE);
    CUDATensorWrapper tw(input);
    kernel_func<<<grid_size, BLOCK_SIZE>>>(tw, tw);
    CHECK(hipGetLastError());
}


Tensor _elementwise_activation(const Tensor & input, void kernel_func(CUDATensorWrapper, CUDATensorWrapper))
{
    assert(input.device == Device::CUDA);
    Tensor output = Tensor::zeros_like(input);
    int grid_size = setup_grid_size(input.numel(), BLOCK_SIZE);
    CUDATensorWrapper itw(input);
    CUDATensorWrapper otw(output);
    kernel_func<<<grid_size, BLOCK_SIZE>>>(itw, otw);
    CHECK(hipGetLastError());
    return std::move(output);
}


Tensor _elementwise_activation_backward(
    const Tensor & grad,
    const Tensor & input,
    void kernel_func(CUDATensorWrapper, CUDATensorWrapper, CUDATensorWrapper))
{
    assert(grad.device == Device::CUDA);    
    assert(input.device == Device::CUDA);

    Tensor output = Tensor::zeros_like(grad);
    int grid_size = setup_grid_size(input.numel(), BLOCK_SIZE);
    CUDATensorWrapper itw(input);
    CUDATensorWrapper otw(output);
    CUDATensorWrapper gtw(grad);
    kernel_func<<<grid_size, BLOCK_SIZE>>>(gtw, itw, otw);
    CHECK(hipGetLastError());
    return std::move(output);
}


void relu_(Tensor & input)
{
    _elementwise_activation_inplace(input, kerner_relu);
}


Tensor relu(const Tensor & input)
{
    return _elementwise_activation(input, kerner_relu);
}


void sigmoid_(Tensor & input)
{
    _elementwise_activation_inplace(input, kerner_sigmoid);
}


Tensor sigmoid(const Tensor & input)
{
    return _elementwise_activation(input, kerner_sigmoid);
}


void softmax_(Tensor & input, size_t dim)
{
    assert(dim < 4);
    assert(input.device == Device::CUDA);    
    int grid_size = setup_grid_size(input.numel(), BLOCK_SIZE);
    CUDATensorWrapper tw(input);

    auto buffer = Tensor::zeros(1, 1, 1, 1, Device::CUDA);
    CUDATensorWrapper buffertw(buffer);
    kerner_softmax<<<grid_size, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(tw, dim, tw, buffertw);
    CHECK(hipGetLastError());
}


Tensor softmax(const Tensor & input, size_t dim)
{
    assert(dim < 4);
    return input;    
}


Tensor ReLU::forward(const Tensor & t)
{    
    _context.clear();
    _context.push_back(t);
    return relu(t);
}


Tensor ReLU::backward(const Tensor & grad)
{   
    assert(!_context.empty());
    auto input = _context[0];
    return _elementwise_activation_backward(grad, input, kerner_relu_backward);
}


Tensor Sigmoid::forward(const Tensor & t)
{
    _context.clear();
    _context.push_back(t);
    return sigmoid(t);
}


Tensor Sigmoid::backward(const Tensor & grad)
{   
    assert(!_context.empty());
    auto input = _context[0];
    return _elementwise_activation_backward(grad, input, kerner_sigmoid_backward);
}

}