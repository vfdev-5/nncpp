#include "hip/hip_runtime.h"
// STD
#include <iostream>

// NNCpp
#include "tensor.hpp"


namespace nncpp
{


void _unifiedmem_free(float * data)
{
    hipFree(data);
}


void _unifiedmem_init(float ** data, size_t n, size_t c, size_t h, size_t w, float init_value)
{
    size_t size = n * c * h * w;
    hipMallocManaged(data, size * sizeof(float));
    for (size_t i=0; i < size; i++)
    {
        (*data)[i] = init_value;
    }
}

void _unifiedmem_zeros(float ** data, size_t n, size_t c, size_t h, size_t w)
{   
    _unifiedmem_init(data, n, c, h, w, 0.0f);
}


void _unifiedmem_ones(float ** data, size_t n, size_t c, size_t h, size_t w)
{   
    _unifiedmem_init(data, n, c, h, w, 1.0f);
}

void _cuda_sync()
{
    hipDeviceSynchronize();
}


}


std::ostream& operator<<(std::ostream& os, nncpp::Tensor & t)
{
    std::string device = "CPU";
    if (t.device == nncpp::Device::CUDA)
    {
        hipDeviceSynchronize();
        device = "CUDA";
    }    
    os << "Tensor: on " << device << ", (" << t.shape[0] << ", " << t.shape[1] << ", " << t.shape[2] << ", " << t.shape[3] << ")"<< std::endl;
    os << "[ " << std::endl;
    for (size_t i = 0; i < std::min(t.shape[0], (size_t) 10); i++)
    {
        os << "  [ ";
        for (size_t j = 0; j < std::min(t.shape[1], (size_t) 10); j++)
        {
            os << "[";
            for (size_t k = 0; k < std::min(t.shape[2], (size_t) 10); k++)
            {
                os << "[";
                for (size_t l = 0; l < std::min(t.shape[3], (size_t) 10); l++)
                {   
                    os << (float) t.at(i, j, k, l) << " ";
                }
                os << "]";
            }
            os << "] ";
        }
        os << "]" << std::endl;
    }
    os << "]" << std::endl;    
    return os;
}
